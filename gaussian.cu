#include "hip/hip_runtime.h"
// C++ prgroam to generate Gaussian filter 
#include <cmath> 
#include <iomanip> 
#include <iostream> 
#include "hip/hip_runtime.h"

#include "opencv2/opencv.hpp"
#include "opencv2/core.hpp"
#include <opencv2/cudaarithm.hpp>

void check(hipError_t x) {
    fprintf(stderr, "%s\n", hipGetErrorString(x));
}

void showMatrix2(double* v1, int width, int height) {
    printf("---------------------\n");
    for (int i = 0; i < width; i++) {
        for (int j = 0; j < height; j++) {
            printf("%8.6lf ", v1[i * width + j]);
        }
        printf("\n");
    }
}

// Function to create Gaussian filter 
void FilterCreation(double* gaussian, int dim) 
{ 
    // intialising standard deviation to 1.0 
    double sigma = 4.0; 
    double r, s = 2.0 * sigma * sigma; 
    // sum is for normalization 
    double sum = 0.0; 
    // generating dim x dim kernel
    int filterWidth = dim / 2;
    for (int x = -filterWidth; x <= filterWidth; x++) { 
        for (int y = -filterWidth; y <= filterWidth; y++) {
                r = sqrt((float) x * x + y * y); 
                gaussian[(x + filterWidth)*dim + (y + filterWidth)] = (exp(-(r * r) / s)) / (M_PI * s);
                sum += gaussian[(x + filterWidth)*dim + (y + filterWidth)];
        }
    }
    // normalising the Kernel 
    for (int i = 0; i < dim; ++i) { 
        for (int j = 0; j < dim; ++j) { 
            gaussian[i*dim + j] /= sum; 
        }
    }
}

__global__ 
void kernel(double* tab, double* gaussian, int width, int height, int pitch) {
	//x_offset = threadIdx.x + blockIdx.x * blockDim.x;
	//y_offset = threadIdx.y + blockIdx.y * blockDim.y;
	//tab[x0+x_offset + (y0+y_offset)*image_width] = 1
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    if (row < width && col < height) {
        //*( ((double *)(((char *)tab) + (row * pitch))) + col) = 1.0f;
		tab[row * pitch + col] = 1.0f;
    }
}
  
// Driver program to test above function 
int main() 
{

    size_t pitch;
    int N = 1<<20;
    int blockSize = 512;
    int numBlocks = (N + blockSize - 1) / blockSize;

    // Create Gaussian circle
    int dim = 15;
    double* GKernel;
	double* DKernel;
    int filterSize = dim * dim * sizeof(double);
    check(hipMallocPitch(&GKernel, &pitch, dim * sizeof(double), dim));
	DKernel = (double*)malloc(filterSize);
    check(hipMemset(GKernel, 0, filterSize));
    FilterCreation(DKernel, dim); 
	//check(hipMemcpy2D(DKernel, dim*sizeof(double), GKernel, pitch, dim*sizeof(double), dim, hipMemcpyDeviceToHost));
    showMatrix2(DKernel, dim, dim);

	int imgSize = 16;
    double* d_tab;
	double* h_tab;
    dim3 grid(4,4);
	dim3 block(4,4); 
    int realSize = imgSize * imgSize * sizeof(double);
    check(hipMallocPitch(&d_tab, &pitch, imgSize * sizeof(double), imgSize));
    h_tab = (double*)malloc(realSize);
    check( hipMemset(d_tab, 0, realSize) );
    kernel <<<grid, block>>>(d_tab, GKernel, imgSize, imgSize, pitch);
    check(hipMemcpy2D(h_tab, imgSize*sizeof(double), d_tab, pitch, imgSize*sizeof(double), imgSize, hipMemcpyDeviceToHost));
    showMatrix2(h_tab, imgSize, imgSize);
    printf("\nPitch size: %d \n", pitch);
    getchar();

    // Read blank image and display
    uint8_t *imgPtr;
    cv::Mat srcImg, dstImg;
    cv::cuda::GpuMat gpuImg;

    srcImg = cv::imread("figs/blank.jpg", cv::IMREAD_GRAYSCALE);
    gpuImg.upload(srcImg);
    hipMalloc((void **)&imgPtr, gpuImg.rows*gpuImg.step);
    check(hipMemcpyAsync(imgPtr, gpuImg.ptr<uint8_t>(), gpuImg.rows*gpuImg.step, hipMemcpyDeviceToDevice));
    cv::cuda::GpuMat gpuSrc(srcImg.rows, srcImg.cols, srcImg.type(), imgPtr, gpuImg.step);
    gpuImg.download(dstImg);
    // cv::imshow("test", dstImg);
    // cv::waitKey(0);
    cv::imwrite("figs/blank_mod.png", dstImg);

    return 0;
}
